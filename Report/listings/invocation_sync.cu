for (int i = 0; i < nStreams; i++)
{
    hipMemcpy(d_A[i], h_A[i], mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B[i], h_B[i], mem_size_B, hipMemcpyHostToDevice);

    matrixMulCUDA<block_size><<<grid, threads>>>(d_C[i], d_A[i], d_B[i], dimsA.x, dimsB.x);
    
    hipMemcpy(h_C[i], d_C[i], mem_size_C, hipMemcpyDeviceToHost);
}