hipStream_t stream[nStreams];
for (int i = 0; i < nStreams; ++i)
    hipStreamCreate(&stream[i]);

for (int i = 0; i < nStreams; ++i)
{
    hipMemcpyAsync(d_A[i], h_A[i], mem_size_A, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync(d_B[i], h_B[i], mem_size_B, hipMemcpyHostToDevice, stream[i]);
}

for (int i = 0; i < nStreams; ++i)
    matrixMulCUDA<block_size><<<grid, threads, 0, stream[i]>>>(d_C[i], d_A[i], d_B[i], dimsA.x, dimsB.x);

for (int i = 0; i < nStreams; ++i)
    hipMemcpyAsync(h_C[i], d_C[i], mem_size_C, hipMemcpyDeviceToHost, stream[i]);